#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void fused_linear_relu_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ weight,
    const scalar_t* __restrict__ bias,
    scalar_t* __restrict__ output,
    int M, int N, int K) {

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < M && col < N) {
    scalar_t acc = 0;
    for (int i = 0; i < K; ++i) {
      acc += input[row * K + i] * weight[col * K + i];
    }
    acc += bias[col];
    output[row * N + col] = acc > 0 ? acc : 0;  // ReLU
  }
}